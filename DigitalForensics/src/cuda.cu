#include "hip/hip_runtime.h"
#include <chrono>
#include <cstring>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <map>
#include <queue>
#include <sstream>
#include <vector>

#include "../include/base.hpp"
#include "../include/read_helper.hpp"
#include "../include/vector_builder.hpp"

__global__ void search_pattern_cuda(const Node *const vec, const char *text,
                                    int text_length, int *indices,
                                    unsigned int *sizes, int max_size,
                                    unsigned int *locks, int thread_count,
                                    int max_pattern_length)
{

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    int start_index = (text_length / thread_count) * tid;
    int end_index =
        start_index + (text_length / thread_count) + max_pattern_length - 2;
    if (end_index >= text_length)
    {
        end_index = text_length - 1;
    }

    if (tid == 0)
    {
        start_index = 0;
    }

    int cur = 0;
    for (int i = start_index; i <= end_index; i++)
    {
        int c = static_cast<unsigned char>(text[i]);
        int pos = c - (c / 32) * 32;
        if (vec[cur].bitmap[c / 32] & (1 << pos))
        {

            // Apply the mask to the number to zero out bits beyond the i-th bit
            unsigned int mask = ((1LL << (pos + 1LL)) - 1LL);
            unsigned int maskedNumber = vec[cur].bitmap[c / 32] & mask;

            int bits_before = 0;
            for (int j = 0; j < (c / 32); ++j)
            {
                bits_before += __popc(vec[cur].bitmap[j]);
            }
            // Move to child node
            cur = vec[cur].offset + bits_before + __popc(maskedNumber) - 1;

            int overlap = ((start_index != 0) &&
                           (i - start_index + 1 < max_pattern_length));

            if (overlap)
            {
                continue;
            }

            if ((vec[cur].pattern_idx >= 0))
            {

                int pattern_index = vec[cur].pattern_idx;
                // Locking
                bool leaveLoop = false;
                while (!leaveLoop)
                {
                    if (atomicExch(&locks[pattern_index], 1u) != 0u)
                    {
                        // critical section
                        unsigned int idx =
                            atomicInc(&sizes[pattern_index], max_size);
                        atomicExch(&indices[pattern_index * max_size + idx], i);
                        leaveLoop = true;
                        // unlocking
                        atomicExch(&locks[pattern_index], 0u);
                    }
                }
            }

            int temp = vec[cur].output_link;
            while (temp != 0)
            {
                int output_pattern_index = vec[temp].pattern_idx;
                // Locking
                bool leaveLoop = false;
                while (!leaveLoop)
                {
                    if (atomicExch(&locks[output_pattern_index], 1u) != 0u)
                    {
                        //  Critical section
                        unsigned int idx =
                            atomicInc(&sizes[output_pattern_index], max_size);
                        atomicExch(
                            &indices[output_pattern_index * max_size + idx], i);
                        leaveLoop = true;
                        // Unlocking
                        atomicExch(&locks[output_pattern_index], 0u);
                    }
                }
                temp = vec[temp].output_link;
            }
        }
        else
        {
            while (cur != 0 && ((vec[cur].bitmap[c / 32] & (1 << pos)) == 0))
            {
                cur = vec[cur].fail_link;
            }

            if ((vec[cur].bitmap[c / 32] & (1 << pos)))
                i--;
        }
    }
}

int main(int argc, char *argv[])
{

    if (argc != 3 || (argc > 1 && std::string(argv[1]) == "--help"))
    {
        std::cout << "Usage: " << argv[0] << " <input_file> <patterns_file>\n";
        std::cout << "  <input_file> : The name of the text file to process.\n";
        std::cout << "  <patterns_file> : The file containing patterns to "
                     "search within the input file.\n";
        return 1;
    }

    std::string inputFileName = argv[1];
    std::string patternsFileName = argv[2];

    const std::string isoFilePath = inputFileName;
    std::string text = readISOFileAsBytes(isoFilePath);

    int text_length = text.length();
    std::cout << "File size in bytes: " << text.size() << std::endl;

    int k = 4; // Number of patterns
    std::string filename = patternsFileName;
    std::vector<std::string> patterns = readHexFile(filename, k);

    int max_pattern_length = 0;
    for (int i = 0; i < k; ++i)
    {
        max_pattern_length = max(max_pattern_length, (int)(patterns[i].size()));
    }

    size_t size = 1;
    node *root = add_node();

    build_automata(root, patterns, size);

    build_failure_output_links(root);

    Node *vec = new Node[size];
    memset(vec, 0, sizeof(Node) * size);

    std::cout << "Nr. patterns = " << k << " Nr. of nodes : " << size
              << " Size : " << size * sizeof(Node) << '\n';

    size_t next_idx = 1;
    flatten_tree(root, vec, size, 0, next_idx);

    // Host pointers
    unsigned int *h_sizes = new unsigned int[k]();
    int max_size = 100; // Maximum number of occurrences per pattern
    int *h_indices = new int[k * max_size];

    // Device pointers
    Node *d_vec;
    char *d_text;
    int *d_indices;
    unsigned int *d_sizes;

    // Allocate memory on the device
    hipMalloc(&d_vec, sizeof(Node) * size);
    hipMalloc(&d_text, sizeof(char) * text_length);
    hipMalloc(&d_sizes, sizeof(unsigned int) * k);
    hipMalloc(&d_indices, sizeof(int) * k * max_size);

    // Copy data from host to device
    hipMemcpy(d_text, text.c_str(), sizeof(char) * text_length,
               hipMemcpyHostToDevice);
    hipMemcpy(d_sizes, h_sizes, sizeof(unsigned int) * k,
               hipMemcpyHostToDevice);
    hipMemcpy(d_vec, vec, sizeof(Node) * size, hipMemcpyHostToDevice);

    unsigned int *d_locks;
    hipMalloc(&d_locks, sizeof(unsigned int) * k);
    hipMemset(d_locks, 0, sizeof(unsigned int) * k);

    int thread_count = 4;
    int block_count = 1;
    auto start = std::chrono::high_resolution_clock::now();

    search_pattern_cuda<<<block_count, thread_count>>>(
        d_vec, d_text, text_length, d_indices, d_sizes, max_size, d_locks,
        thread_count * block_count, max_pattern_length);

    // Synchronize after kernel launch to catch any errors during execution
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();

    // Copy results back to host
    hipMemcpy(h_indices, d_indices, sizeof(int) * k * max_size,
               hipMemcpyDeviceToHost);
    hipMemcpy(h_sizes, d_sizes, sizeof(unsigned int) * k,
               hipMemcpyDeviceToHost);

    // Duration in milliseconds
    auto duration =
        std::chrono::duration_cast<std::chrono::milliseconds>(end - start);

    std::cout << "Time taken: " << duration.count() << " milliseconds"
              << std::endl;

    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::cout << "CUDA error: " << hipGetErrorString(error) << std::endl;
        return -1;
    }

    error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::cout << "CUDA error after synchronize: "
                  << hipGetErrorString(error) << std::endl;
        return -1;
    }

    // Print results
    for (int i = 0; i < patterns.size(); i++)
    {
        std::cout << "Total occurrences of \"" << patterns[i];
        std::cout << "\": " << h_sizes[i] << std::endl;
        std::cout << "Positions: ";
        for (int j = 0; j < h_sizes[i]; j++)
        {
            std::cout << h_indices[i * max_size + j] << " ";
        }

        std::cout << std::endl;
    }

    // Free device memory
    hipFree(d_indices);
    hipFree(d_sizes);
    hipFree(d_text);
    hipFree(d_vec);
    // After kernel execution
    hipFree(d_locks);
    delete[] h_indices;
    delete[] h_sizes;

    return 0;
}